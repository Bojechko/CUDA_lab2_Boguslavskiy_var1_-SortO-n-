#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <math.h>
#include <stdio.h>
#include <iostream>
using namespace std;


#define N (100)
#define k (20)
const int threadsPerBlock = 1024;



 __global__ void sortKernel(int* dev_arr, int* helper)
    {
        int schet = 0;
        int powtor = 0;
        __shared__ int temp[1024];
        if (blockIdx.x * blockDim.x + threadIdx.x < N)
        {
            temp[threadIdx.x] = dev_arr[blockIdx.x * blockDim.x + threadIdx.x];
            for (int i = 0; i < N; ++i) {

                if (dev_arr[i] < temp[threadIdx.x])
                    ++schet; //позиция в результате

                if (dev_arr[i] == temp[threadIdx.x])
                    ++powtor; //позиция в результате
            }

            helper[dev_arr[blockIdx.x * blockDim.x + threadIdx.x]] = powtor;


            // dev_arr[schet] = temp[threadIdx.x];
        }
    

}


 __global__ void sortKernelFinal(int* dev_arr, int* helper)
 {
     

       int b = 0;
       for (int i = 0; i < k + 1; ++i) {
           for (int j = 0; j < helper[i]; ++j) {
               dev_arr[b++] = i;
           }
       }
      // __syncthreads();*/




 }
__global__ void get_arr(int* dev_arr) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = 0;
    
    if (idx < N) {
        if (idx < k-2)
        {

            if (idx % 2 == 0)
                dev_arr[idx] = idx;
            else
                dev_arr[idx] = idx + 2;
        } else
            dev_arr[idx] = idx % k;

        printf(" %d, ", dev_arr[idx]);
    }

 /*   if (idx < N) {
        dev_arr[idx] = N - idx;
        printf(" %d, ", dev_arr[idx]);
    }*/
    //

}

__global__ void get_arr_zero(int* dev_zero) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    if (idx < k) {   

        dev_zero[idx] = 0;
        
    }
    //printf(" %d, ", dev_zero[idx]);

}


__global__ void show_arr(int* dev_arr) {
    __syncthreads();
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf(" %d, ", dev_arr[idx]);

}

void get_array_for_CPU(int* mas) {
    for (int i = 0; i < N; i++) {
        mas[i] = rand() % k;;
    }
    for (int i = 0; i < N - 1; i++)
        printf(" %d, ", mas[i]);
    printf("\n");

}
void sort_for_CPU(int* mas, int* masHelper)
{
    for (int i = 0; i < N - 1; ++i) {
        ++masHelper[mas[i]];
    }

    int b = 0;
    for (int i = 0; i < k + 1; ++i) {
        for (int j = 0; j < masHelper[i]; ++j) {
            mas[b++] = i;
        }
    }
    printf("after sort \n");
    for (int i = 0; i < N-1; i++)
        printf(" %d, ", mas[i]);
    
}


int main() {
    //GPU
    int* host_arr = new int[N];
    int* dev_arr = new int[N];
  //  int* dev_res = new int[N];
    int* dev_help = new int[k];
    float elapsedTimeInMs = 0.0f;


    hipDeviceReset();


    hipMalloc((void**)&dev_arr, N * sizeof(int));
  //  hipMalloc((void**)&dev_res, N * sizeof(int));
    hipMalloc((void**)&dev_help, N * sizeof(int));

    get_arr << <dim3(((N + 511) / 512), 1), dim3(threadsPerBlock, 1) >> > (dev_arr);
   // get_arr_zero << <dim3(((N + 511) / 512), 1), dim3(threadsPerBlock, 1) >> > (dev_help);

   // printf(" -------------------------------------------- \n");
    //show_arr << <dim3(((N + 511) / 512), 1), dim3(threadsPerBlock, 1) >> > (dev_arr);
    printf("\n");
    printf(" -------------------------------------------- \n");
    printf(" GPU \n");
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipEventSynchronize(start);
    hipDeviceSynchronize();
    sortKernel << <dim3(((N + 511) / 512), 1), dim3(threadsPerBlock, 1) >> > (dev_arr, dev_help);
    sortKernelFinal << <dim3(((N + 511) / 512), 1), dim3(threadsPerBlock, 1) >> > (dev_arr, dev_help);
    hipDeviceSynchronize();
    hipMemcpy(host_arr, dev_arr, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTimeInMs, start, stop);
    
    printf("  \n");
    printf(" -------------------------------------------- \n");
    for (int i = 0; i < N - 1; i++)
        printf(" %d, ", host_arr[i]);
    //for (int i = 0; i < N; i++)
      //  printf("=>%d", host_arr[i]);
    hipFree(dev_arr);
    delete[]host_arr;

    printf("Time in GPU %f\n", elapsedTimeInMs / 1000);
   
    printf(" -------------------------------------------- \n");
    printf(" -------------------------------------------- \n");
    printf(" CPU \n");
    printf(" -------------------------------------------- \n");
    printf(" -------------------------------------------- \n");
    //CPU
    int* a = new int[N];
    int masHelper[k] = { 0 };
    clock_t start2;
    double time2;
    start2 = clock();
    
    get_array_for_CPU(a);

    
    sort_for_CPU(a, masHelper);
    
    time2 = (double)(clock() - start2) / CLOCKS_PER_SEC;
    printf("Time in CPU %f\n", time2);




    return 0;
}