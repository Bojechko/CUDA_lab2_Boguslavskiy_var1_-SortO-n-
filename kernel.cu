#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <math.h>
#include <stdio.h>
#include <iostream>
using namespace std;


#define N (20000)
#define k (50)
const int threadsPerBlock = 1024;
//const int threads_qty = N / 2;



__global__ void sortKernel(int* dev_arr)
{
    int schet = 0;
    __shared__ int temp[1024];
    temp[threadIdx.x] = dev_arr[blockIdx.x * blockDim.x + threadIdx.x];
    for (int i = 0; i < N; ++i) {
        if (dev_arr[i] < temp[threadIdx.x])
            ++schet; //позиция в результате
    }

    __syncthreads();
    dev_arr[schet] = temp[threadIdx.x];

}
__global__ void get_arr(int* dev_arr) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        if (idx % 2 == 0)
            dev_arr[idx] = idx;
        else
            dev_arr[idx] = idx + 9;
    }
}


void get_array_for_CPU(int* arr) {
    for (int i = 0; i < N; i++) {
        arr[i] = rand() % k;;
    }
}
void sort_for_CPU(int* mas, int* masHelper)
{
    for (int i = 0; i < N - 1; ++i) {
        ++masHelper[mas[i]];
    }

    int b = 0;
    for (int i = 0; i < k + 1; ++i) {
        for (int j = 0; j < masHelper[i]; ++j) {
            mas[b++] = i;
        }
    }
}


int main() {
    //GPU
    int* host_arr = new int[N];
    int* dev_arr = new int[N];

    float elapsedTimeInMs = 0.0f;


    hipDeviceReset();


    hipMalloc((void**)&dev_arr, N * sizeof(int));

    get_arr << <dim3(((N + 511) / 512), 1), dim3(threadsPerBlock, 1) >> > (dev_arr);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipEventSynchronize(start);
    hipDeviceSynchronize();
    sortKernel << <dim3(((N + 511) / 512), 1), dim3(threadsPerBlock, 1) >> > (dev_arr);
    hipDeviceSynchronize();
    hipMemcpy(host_arr, dev_arr, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
//for (int i = 0; i < N; i++)
  //      printf(host_arr[i]);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTimeInMs, start, stop);
    printf("Time in GPU %f\n", elapsedTimeInMs / 1000);
    

    hipFree(dev_arr);
    delete[]host_arr;

    //CPU
    int* a = new int[N];
    int masHelper[k] = { 0 };
    clock_t start2;
    double time2;
    start2 = clock();

    get_array_for_CPU(a);
    sort_for_CPU(a, masHelper);

    time2 = (double)(clock() - start2) / CLOCKS_PER_SEC;
    printf("Time in CPU %f\n", time2);




    return 0;
}